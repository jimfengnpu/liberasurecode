#include "hip/hip_runtime.h"

/*
 * =====================================================================================
 *
 *       Filename:  decode.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/2012 10:50:55 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (yszheda AT gmail.com),
 *        Company:  
 *
 * =====================================================================================
 */

#include "decode.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "matrix.h"
#include "cpu_decode.h"


struct ThreadDataType {
    int id;
    int nativeBlockNum;
    int parityBlockNum;
    int chunkSize;
    int totalSize;
    int gridDimXSize;
    int streamNum;
    uint8_t* dataBuf;
    uint8_t* codeBuf;
    uint8_t* decodingMatrix;
};	/* ----------  end of struct ThreadDataType  ---------- */

typedef struct ThreadDataType ThreadDataType;

static pthread_barrier_t barrier;

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  show_square_matrix_debug
 *  Description:  show the content of a square matrix
 *  Used only for debugging
 * =====================================================================================
 */
#ifdef DEBUG
void show_squre_matrix_debug(uint8_t *matrix, int size)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%d ", matrix[i*size+j]);
        }
        printf("\n");
    }
}
#endif
/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  copy_matrix
 *  Description:  copy the row with <srcRowIndex> from the matrix <src>
 *  to the row with <desRowIndex> of the matrix <des>
 * =====================================================================================
 */
void copy_matrix(uint8_t *src, uint8_t *des, int srcRowIndex, int desRowIndex, int rowSize)
{
    for (int i = 0; i < rowSize; i++)
    {
        des[desRowIndex * rowSize + i] = src[srcRowIndex * rowSize + i];
    }
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  decode
 *  Description:  decode the given buffer of code chunks in the GPU with <id>
 * =====================================================================================
 */
void decode(uint8_t *dataBuf, uint8_t *codeBuf, uint8_t *decodingMatrix, int id, int nativeBlockNum, int parityBlockNum, int chunkSize, int gridDimXSize, int streamNum)
{
    float totalCommunicationTime = 0;
    // compute total execution time
    float totalTime;
    hipEvent_t totalStart, totalStop;
    // create event
    checkCudaErrors(hipEventCreate(&totalStart));
    checkCudaErrors(hipEventCreate(&totalStop));
    checkCudaErrors(hipEventRecord(totalStart));

    // compute step execution time
    float stepTime;
    hipEvent_t stepStart, stepStop;
    // create event
    checkCudaErrors(hipEventCreate(&stepStart));
    checkCudaErrors(hipEventCreate(&stepStop));

    int matrixSize = nativeBlockNum * nativeBlockNum * sizeof(uint8_t);
    uint8_t *decodingMatrix_d;	//device
    checkCudaErrors(hipMalloc((void **) &decodingMatrix_d, matrixSize));

    // record event
    checkCudaErrors(hipEventRecord(stepStart));
    checkCudaErrors(hipMemcpy(decodingMatrix_d, decodingMatrix, matrixSize, hipMemcpyHostToDevice));
    // record event and synchronize
    checkCudaErrors(hipEventRecord(stepStop));
    checkCudaErrors(hipEventSynchronize(stepStop));
    // get event elapsed time
    checkCudaErrors(hipEventElapsedTime(&stepTime, stepStart, stepStop));
    printf("Device%d: Copy decoding matrix from CPU to GPU: %fms\n", id, stepTime);
    totalCommunicationTime += stepTime;

    // NOTE: use CUDA stream to decode the file
    // to achieve computation and comunication overlapping
    // Use DFS way
    int streamMinChunkSize = chunkSize / streamNum;
    hipStream_t stream[streamNum];
    for (int i = 0; i < streamNum; i++)
    {
        checkCudaErrors(hipStreamCreate(&stream[i]));
    }

    uint8_t *dataBuf_d[streamNum];		//device
    uint8_t *codeBuf_d[streamNum];		//device
    for (int i = 0; i < streamNum; i++)
    {
        int streamChunkSize = streamMinChunkSize;
        if (i == streamNum - 1)
        {
            streamChunkSize = chunkSize - i * streamMinChunkSize;
        }

        int dataSize = nativeBlockNum * streamChunkSize * sizeof(uint8_t);
        int codeSize = nativeBlockNum * streamChunkSize * sizeof(uint8_t);

        checkCudaErrors(hipMalloc((void **)&dataBuf_d[i], dataSize));
        checkCudaErrors(hipMalloc((void **)&codeBuf_d[i], codeSize));
    }

    for (int i = 0; i < streamNum; i++)
    {
        int streamChunkSize = streamMinChunkSize;
        if (i == streamNum - 1)
        {
            streamChunkSize = chunkSize - i * streamMinChunkSize;
        }

        for (int j = 0; j < nativeBlockNum; j++)
        {
            checkCudaErrors(hipMemcpyAsync(codeBuf_d[i] + j * streamChunkSize,
                    codeBuf + j * chunkSize + i * streamMinChunkSize,
                    streamChunkSize * sizeof(uint8_t),
                    hipMemcpyHostToDevice,
                    stream[i]));
        }

        stepTime = decode_chunk(dataBuf_d[i], decodingMatrix_d, codeBuf_d[i], nativeBlockNum, parityBlockNum, streamChunkSize, gridDimXSize, stream[i]);

        for (int j = 0; j < nativeBlockNum; j++)
        {
            checkCudaErrors(hipMemcpyAsync(dataBuf + j * chunkSize + i * streamMinChunkSize,
                    dataBuf_d[i] + j * streamChunkSize,
                    streamChunkSize * sizeof(uint8_t),
                    hipMemcpyDeviceToHost,
                    stream[i]));
        }
    }

    for (int i = 0; i < streamNum; i++)
    {
        checkCudaErrors(hipFree(dataBuf_d[i]));
        checkCudaErrors(hipFree(codeBuf_d[i]));
    }
    checkCudaErrors(hipFree(decodingMatrix_d));

    // record event and synchronize
    checkCudaErrors(hipEventRecord(totalStop));
    checkCudaErrors(hipEventSynchronize(totalStop));
    // get event elapsed time
    checkCudaErrors(hipEventElapsedTime(&totalTime, totalStart, totalStop));
    printf("Device%d: Total GPU decoding time: %fms\n", id, totalTime);

    for(int i = 0; i < streamNum; i++)
    {
        checkCudaErrors(hipStreamDestroy(stream[i]));
    }
}

static void* GPU_thread_func(void * args)
{
    ThreadDataType* thread_data = (ThreadDataType *) args;
    checkCudaErrors(hipSetDevice(thread_data->id));

    struct timespec start, end;
    pthread_barrier_wait(&barrier);
    clock_gettime(CLOCK_REALTIME, &start);
    pthread_barrier_wait(&barrier);

    decode(thread_data->dataBuf,
            thread_data->codeBuf,
            thread_data->decodingMatrix,
            thread_data->id,
            thread_data->nativeBlockNum,
            thread_data->parityBlockNum,
            thread_data->chunkSize,
            thread_data->gridDimXSize,
            thread_data->streamNum);

    pthread_barrier_wait(&barrier);
    clock_gettime(CLOCK_REALTIME, &end);
    if (thread_data->id == 0)
    {
        double totalTime = (double) (end.tv_sec - start.tv_sec) * 1000
            + (double) (end.tv_nsec - start.tv_nsec) / (double) 1000000L;
        printf("Total GPU decoding time using multiple devices: %fms\n", totalTime);
    }
    return NULL;
}

/* 
 * ===  FUNCTION  ======================================================================
 *         Name:  decode_file
 *  Description:  decode the original input file <fileName> with the given settings
 * =====================================================================================
 */
extern "C"
void decode_data(uint8_t *generatorMatrix, uint8_t **data, uint8_t **parity, int *missing, int nativeBlockNum, int parityBlockNum, int chunkSize, int gridDimXSize, int streamNum)
{

    int matrixSize;
    uint8_t *encodingMatrix;	//host
    uint8_t **availableData;

    matrixSize = nativeBlockNum * nativeBlockNum;
    encodingMatrix = (uint8_t*) malloc(matrixSize);
    availableData = (uint8_t**) malloc(sizeof(uint8_t*) * nativeBlockNum);
    // NOTE: Pinned host memory is expensive for allocation,
    // so pageable host memory is used here.

    int i = 0;

    for (int index = 0; index < nativeBlockNum + parityBlockNum && i < nativeBlockNum; index++)
    {
        if(!missing[index]) {
            availableData[i] = (index < nativeBlockNum)? data[index]: parity[index - nativeBlockNum];
            copy_matrix(generatorMatrix, encodingMatrix, index, i, nativeBlockNum);
            i++;
        }
    }

    hipDeviceProp_t deviceProperties;
    checkCudaErrors(hipGetDeviceProperties(&deviceProperties, 0));
    int maxGridDimXSize = min(deviceProperties.maxGridSize[0], deviceProperties.maxGridSize[1]);
    if (gridDimXSize > maxGridDimXSize || gridDimXSize <= 0)
    {
        printf("Valid grid size: (0, %d]\n", maxGridDimXSize);
        gridDimXSize = maxGridDimXSize;
    }

    uint8_t *decodingMatrix;
    // Pageable Host Memory is preferred here since the decodingMatrix is small
    decodingMatrix = (uint8_t*) malloc(matrixSize);
    CPU_invert_matrix(encodingMatrix, decodingMatrix, nativeBlockNum);

    int GPU_num;
    checkCudaErrors(hipGetDeviceCount(&GPU_num));

    void* threads = malloc(GPU_num * sizeof(pthread_t));
    ThreadDataType* thread_data = (ThreadDataType *) malloc(GPU_num * sizeof(ThreadDataType));

    uint8_t *dataBufPerDevice[GPU_num];
    uint8_t *codeBufPerDevice[GPU_num];
    pthread_barrier_init(&barrier, NULL, GPU_num);

    int minChunkSizePerDevice = chunkSize / GPU_num;
    for (int i = 0; i < GPU_num; ++i)
    {
        checkCudaErrors(hipSetDevice(i));

        thread_data[i].id = i;
        thread_data[i].nativeBlockNum = nativeBlockNum;
        thread_data[i].parityBlockNum = parityBlockNum;
        int deviceChunkSize = minChunkSizePerDevice;
        if (i == GPU_num - 1)
        {
            deviceChunkSize = chunkSize - i * minChunkSizePerDevice;
        }
        thread_data[i].chunkSize = deviceChunkSize;
        thread_data[i].gridDimXSize = gridDimXSize;
        thread_data[i].streamNum = streamNum;
        int deviceDataSize = nativeBlockNum * deviceChunkSize * sizeof(uint8_t);
        int deviceCodeSize = nativeBlockNum * deviceChunkSize * sizeof(uint8_t);
        checkCudaErrors(hipHostMalloc((void **)&dataBufPerDevice[i], deviceDataSize));
        checkCudaErrors(hipHostMalloc((void **)&codeBufPerDevice[i], deviceCodeSize));
        for (int j = 0; j < nativeBlockNum; ++j)
        {
            // Pinned Host Memory
            checkCudaErrors(hipMemcpy(codeBufPerDevice[i] + j * deviceChunkSize,
                    availableData[j] + i * minChunkSizePerDevice,
                    deviceChunkSize,
                    hipMemcpyHostToHost));
        }
        thread_data[i].dataBuf = dataBufPerDevice[i];
        thread_data[i].codeBuf = codeBufPerDevice[i];
        thread_data[i].decodingMatrix = decodingMatrix;

        pthread_create(&((pthread_t*) threads)[i], NULL, GPU_thread_func, (void *) &thread_data[i]);
    }

    for (int i = 0; i < GPU_num; ++i)
    {
        pthread_join(((pthread_t*) threads)[i], NULL);
    }

    for (int i = 0; i < GPU_num; ++i)
    {
        int deviceChunkSize = minChunkSizePerDevice;
        if (i == GPU_num - 1)
        {
            deviceChunkSize = chunkSize - i * minChunkSizePerDevice;
        }

        for (int j = 0; j < nativeBlockNum; ++j)
        {
            // Pinned Host Memory
            if(missing[j]){
                checkCudaErrors(hipMemcpy(data[j] + i * minChunkSizePerDevice,
                    dataBufPerDevice[i] + j * deviceChunkSize,
                    deviceChunkSize,
                    hipMemcpyHostToHost));
            }
            
        }

        // Pinned Host Memory
        checkCudaErrors(hipHostFree(dataBufPerDevice[i]));
        checkCudaErrors(hipHostFree(codeBufPerDevice[i]));
    }

    pthread_barrier_destroy(&barrier);
    checkCudaErrors(hipDeviceReset());

    // NOTE: Pinned host memory is expensive for deallocation,
    // so pageable host memory is used here.
    free(decodingMatrix);
    free(encodingMatrix);
    free(availableData);
}
